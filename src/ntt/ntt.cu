#include "ntt.cuh"

namespace ntt {
extern "C" hipError_t Ntt(
        uint8_t* input,
        int      input_block_size,
        int      number_of_blocks,
        uint8_t* output) {
    auto config = NttConfig::default_ntt_config();

    return hipError_t::hipSuccess;
}
} // namespace ntt