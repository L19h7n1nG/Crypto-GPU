#include "hip/hip_runtime.h"
#include <cstdint>
#include "Keccak.cuh"

namespace keccak {

template <int C, int D>
__global__ void keccak_hash_kernel(
        uint8_t* input,
        int      input_block_size,
        int      number_of_blocks,
        uint8_t* output) {
    int bid = threadIdx.x + blockDim.x * blockIdx.x;
    if (bid >= number_of_blocks)
        return;

    const int r_bits = 1600 - C;
    const int r_bytes = r_bits / 8;
    const int d_bytes = D / 8;

    uint8_t* b_input = input + bid * input_block_size;
    uint8_t* b_output = output + bid * d_bytes;
    uint64_t state[25] = {};

    int input_len = input_block_size;

    while (input_len >= r_bytes) {
        // #pragma unroll
        for (int i = 0; i < r_bytes; i += 8) {
            state[i / 8] ^= *(uint64_t*)(b_input + i);
        }
        keccakf(state);
        b_input += r_bytes;
        input_len -= r_bytes;
    }

    // last block (if any)
    uint8_t last_block[r_bytes];
    for (int i = 0; i < input_len; i++) {
        last_block[i] = b_input[i];
    }

    // pad 10*1
    last_block[input_len] = 1;
    for (int i = 0; i < r_bytes - input_len - 1; i++) {
        last_block[input_len + i + 1] = 0;
    }
    // last bit
    last_block[r_bytes - 1] |= 0x80;

    // #pragma unroll
    for (int i = 0; i < r_bytes; i += 8) {
        state[i / 8] ^= *(uint64_t*)(last_block + i);
    }
    keccakf(state);

#pragma unroll
    for (int i = 0; i < d_bytes; i += 8) {
        *(uint64_t*)(b_output + i) = state[i / 8];
    }
}

template <int C, int D>
hipError_t keccak_hash(
        uint8_t*            input,
        int                 input_block_size,
        int                 number_of_blocks,
        uint8_t*            output,
        const KeccakConfig& config) {
    CHK_INIT_IF_RETURN();

    auto& stream = config.ctx.stream;

    uint8_t *input_device{nullptr}, *output_device{nullptr};
    if (config.are_inputs_on_device)
        input_device = input;
    else {
        CHK_IF_RETURN(hipMallocAsync(
                &input_device, number_of_blocks * input_block_size, stream));
        CHK_IF_RETURN(hipMemcpyAsync(
                input_device,
                input,
                number_of_blocks * input_block_size,
                hipMemcpyHostToDevice,
                stream));
    }

    if (config.are_outputs_on_device)
        output_device = output;
    else
        CHK_IF_RETURN(hipMallocAsync(
                &output_device, number_of_blocks * (D / 8), stream));

    int number_of_threads = 1024;
    int number_of_gpu_blocks = (number_of_blocks - 1) / number_of_threads + 1;

    keccak_hash_kernel<C, D>
            <<<number_of_gpu_blocks, number_of_threads, 0, stream>>>(
                    input_device,
                    input_block_size,
                    number_of_blocks,
                    output_device);

    if (!config.are_inputs_on_device)
        CHK_IF_RETURN(hipFreeAsync(input_device, stream));

    if (!config.are_outputs_on_device) {
        CHK_IF_RETURN(hipMemcpyAsync(
                output,
                output_device,
                number_of_blocks * (D / 8),
                hipMemcpyDeviceToHost,
                stream));
        CHK_IF_RETURN(hipFreeAsync(output_device, stream));
    }

    if (!config.is_async)
        return CHK_STICKY(hipStreamSynchronize(stream));
    return CHK_LAST();
}

extern "C" hipError_t Keccak256(
        uint8_t* input,
        int      input_block_size,
        int      number_of_blocks,
        uint8_t* output) {
    auto config = KeccakConfig::default_keccak_config();
    return keccak_hash<512, 256>(
            input, input_block_size, number_of_blocks, output, config);
}
} // namespace keccak
