#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime_api.h>

__global__ void myKernel(void) {}

int main(void) {
    myKernel<<<1, 1>>>();
    printf("Hello CUDA!\n");

    std::cout << gpuGetMaxGflopsDeviceId() << '\n';
    return 0;
}